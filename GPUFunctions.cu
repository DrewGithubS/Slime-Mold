
#include <hip/hip_runtime.h>
#include <iostream>

void * gpuMemAlloc(uint32_t bytes) {
	void * output;
	hipError_t err = hipMalloc(&output, bytes);
	if ( err != hipSuccess ) {
		std::cout << hipGetErrorString(err) << std::endl;
		return NULL;
	}

	return output;
};